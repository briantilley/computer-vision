#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "headers/device.h"

// development
#include <iostream>

// #define YUV_TO_RGB_FLOAT

// convert to RGB using ITU 601 standard
__device__ __inline__ // function for GPU thread to use
void YUVtoRGBA(const byte& Y, const byte& U, const byte& V, byte& R, byte& G, byte& B, byte& A)
{
	#ifdef YUV_TO_RGB_FLOAT
		float fY, fU, fV;

		fY = Y * 298.082f / 256.f;
		fU = U / 256.f;
		fV = V / 256.f;

		R = min(max(static_cast<int>(fY + 408.593f * fV - 222.921), 0), 255);
		G = min(max(static_cast<int>(fY - 100.291f * fU - 208.12f * fV + 135.576), 0), 255);
		B = min(max(static_cast<int>(fY + 516.412 * fU - 276.836), 0), 255);
	#else
		int tY = 298 * Y;

		R = min(max((tY + 409 * V - 57068) >> 8, 0), 255);
		G = min(max((tY - 100 * U - 208 * V + 34707 >> 8), 0), 255);
		B = min(max((tY + 516 * U - 70870) >> 8, 0), 255);
	#endif

	// no alpha data in YUV
	A = 255;
}

// set 'clampCoords' to false when grid size
// perfectly corresponds to image size
// (each thread is 8 pixels wide)
template<bool clampCoords=false>
__global__
void kernelNV12toRGBA(const void* const input, const unsigned pitchInput,
					  void* const output, const unsigned pitchOutput,
					  const unsigned pixelsWidth=0, const unsigned pixelsHeight=0)
{
	// make sure we get the right data for clamping coords if necessary
	if(clampCoords)
	{
		if(!(pixelsWidth && pixelsHeight))
			return;
	}

	// // number of rows of threads in use
	// const unsigned gridWidth = gridDim.x * blockDim.x;
	unsigned gridHeight;
	if(clampCoords)
		gridHeight = pixelsHeight;
	else
		gridHeight = gridDim.y * blockDim.y;

	// position within the grid
	// (threads per block) * (position of block in grid) + (position of thread in block)
	const unsigned gridXidx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned gridYidx = blockDim.y * blockIdx.y + threadIdx.y;

	if(clampCoords)
	{
		if(gridXidx * 8 >= pixelsWidth || gridYidx >= pixelsHeight)
			return;
	}

	// NV12 global reads for 8 pixels of data
	// address calculation from inner to outer access:
	// convert to byte array, position to proper row,
	// convert to row array, position to proper column
	word packed_Y_bytes, packed_UV_bytes;
	packed_Y_bytes = reinterpret_cast<const word*>(static_cast<const byte*>(input) + gridYidx * pitchInput)[gridXidx];
	packed_UV_bytes = reinterpret_cast<const word*>(static_cast<const byte*>(input) + (gridHeight + gridYidx / 2) * pitchInput)[gridXidx];

	// array representation of input data
	const byte* const Y  = reinterpret_cast<const byte*>(&packed_Y_bytes);
	const byte* const UV = reinterpret_cast<const byte*>(&packed_UV_bytes);

	// local destination for conversion
	word pixelPairs[4]; // pack into 4 2-pixel/8-byte pairs
	byte* const pair_0_bytes = reinterpret_cast<byte*>(pixelPairs    );
	byte* const pair_1_bytes = reinterpret_cast<byte*>(pixelPairs + 1);
	byte* const pair_2_bytes = reinterpret_cast<byte*>(pixelPairs + 2);
	byte* const pair_3_bytes = reinterpret_cast<byte*>(pixelPairs + 3);

	// this method of computation exposes ILP
	// convert all of the pixel data
	YUVtoRGBA(Y[0], UV[0], UV[1], pair_0_bytes[0], pair_0_bytes[1], pair_0_bytes[2], pair_0_bytes[3]);
	YUVtoRGBA(Y[1], UV[0], UV[1], pair_0_bytes[4], pair_0_bytes[5], pair_0_bytes[6], pair_0_bytes[7]);
	YUVtoRGBA(Y[2], UV[2], UV[3], pair_1_bytes[0], pair_1_bytes[1], pair_1_bytes[2], pair_1_bytes[3]);
	YUVtoRGBA(Y[3], UV[2], UV[3], pair_1_bytes[4], pair_1_bytes[5], pair_1_bytes[6], pair_1_bytes[7]);
	YUVtoRGBA(Y[4], UV[4], UV[5], pair_2_bytes[0], pair_2_bytes[1], pair_2_bytes[2], pair_2_bytes[3]);
	YUVtoRGBA(Y[5], UV[4], UV[5], pair_2_bytes[4], pair_2_bytes[5], pair_2_bytes[6], pair_2_bytes[7]);
	YUVtoRGBA(Y[6], UV[6], UV[7], pair_3_bytes[0], pair_3_bytes[1], pair_3_bytes[2], pair_3_bytes[3]);
	YUVtoRGBA(Y[7], UV[6], UV[7], pair_3_bytes[4], pair_3_bytes[5], pair_3_bytes[6], pair_3_bytes[7]);

	// strided global write of the RGBA data for 8 pixels,
	// taking the hit on efficiency
	word* const row = reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput);
	const unsigned firstColumn = 4 * gridXidx; 

	row[firstColumn    ] = pixelPairs[0];
	row[firstColumn + 1] = pixelPairs[1];
	row[firstColumn + 2] = pixelPairs[2];
	row[firstColumn + 3] = pixelPairs[3];
}

__global__
void kernelNV12toMAG(const void* const input, const unsigned pitchInput,
					 void* const output, const unsigned pitchOutput,
					 const unsigned pixelsWidth, const unsigned pixelsHeight)
{
	// position within the grid
	// (threads per block) * (position of block in grid) + (position of thread in block)
	const unsigned gridXidx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned gridYidx = blockDim.y * blockIdx.y + threadIdx.y;

	if(gridXidx * 8 >= pixelsWidth || gridYidx >= pixelsHeight)
		return;

	// read 8 pixels of luminance data, copy into RGBA format with alpha set to full
	word inputYoctet, outputRGBApairs[4] = { 0xffffffffffffffff, 0xffffffffffffffff, 0xffffffffffffffff, 0xffffffffffffffff };
	inputYoctet = reinterpret_cast<const word*>(static_cast<const byte*>(input) + gridYidx * pitchInput)[gridXidx];

	const byte* const inputBytes = reinterpret_cast<byte*>(&inputYoctet);
	byte* outputBytes = reinterpret_cast<byte*>(outputRGBApairs);

	// copy all 8 pixels of data
	#pragma unroll
	for(int i = 0; i < 8; ++i)
		outputBytes[i * 4] = outputBytes[i * 4 + 1] = outputBytes[i * 4 + 2] = inputBytes[i];

	// strided global write of the RGBA data for 8 pixels,
	// taking the hit on efficiency
	word* const row = reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput);
	const unsigned firstColumn = 4 * gridXidx; 

	row[firstColumn    ] = outputRGBApairs[0];
	row[firstColumn + 1] = outputRGBApairs[1];
	row[firstColumn + 2] = outputRGBApairs[2];
	row[firstColumn + 3] = outputRGBApairs[3];
}

// #define RGB_TO_MAG_FLOAT

__device__ __inline__
void RGBtoMAG(const byte& R, const byte& G, const byte& B, byte& mag)
{
	#ifdef RGB_TO_MAG_FLOAT
		mag = min(static_cast<unsigned>(0.299f * R + 0.587f * G + 0.114f * B) + 0.5f, 255);
	#else
		mag = min((77 * R + 150 * G + 29 * B) >> 8, 255);
	#endif
}

#define RGBA_MAG_GRID_STRIDE_COUNT 2
__global__
void kernelRGBAtoMAG(const void* const input, const unsigned pitchInput,
					 void* const output, const unsigned pitchOutput,
					 const unsigned pixelsWidth, const unsigned pixelsHeight)
{
	// position within grid
	const unsigned gridXidx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned gridYidx = blockDim.y * blockIdx.y + threadIdx.y;

	// used for grid-strided access
	const unsigned activeGridWidth = pixelsWidth / (2 * RGBA_MAG_GRID_STRIDE_COUNT);

	// kill extraneous threads
	if(gridXidx * 2 * RGBA_MAG_GRID_STRIDE_COUNT >= pixelsWidth || gridYidx > pixelsHeight)
		return;

	word pixelPairs[RGBA_MAG_GRID_STRIDE_COUNT];

	// all global reads
	#pragma unroll
	for(int i = 0; i < RGBA_MAG_GRID_STRIDE_COUNT; ++i)
		pixelPairs[i] = reinterpret_cast<const word*>(static_cast<const byte*>(input) + gridYidx * pitchInput)[gridXidx + i * activeGridWidth];

	// conversion
	byte* pixelPairBytes = nullptr;
	#pragma unroll
	for(int i = 0; i < RGBA_MAG_GRID_STRIDE_COUNT; ++i)
	{
		pixelPairBytes = reinterpret_cast<byte*>(pixelPairs + i);

		// conversion (overwrite RGB data)
		RGBtoMAG(pixelPairBytes[0], pixelPairBytes[1], pixelPairBytes[2], pixelPairBytes[0]);
		RGBtoMAG(pixelPairBytes[4], pixelPairBytes[5], pixelPairBytes[6], pixelPairBytes[4]);

		// copy magnitude to other subpixels
		pixelPairBytes[1] = pixelPairBytes[2] = pixelPairBytes[0];
		pixelPairBytes[5] = pixelPairBytes[6] = pixelPairBytes[4];
	}

	// write out new pixels
	#pragma unroll
	for(int i = 0; i < RGBA_MAG_GRID_STRIDE_COUNT; ++i)
		reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput)[gridXidx + i * activeGridWidth] = pixelPairs[i];
}

// 2 pixels per thread
// arrange read and write such that the one memory location can be used
// kernel must be given normalized convolution matrices of odd width and height
template<const int filterMatrixWidth, const int filterMatrixHeight>
__global__
void kernelMatrixConvolution(const void* const input, const int pitchInput,
							 void* const output, const int pitchOutput,
							 const int pixelsWidth, const int pixelsHeight,
							 const float* const filterMatrix)
{
	// // dimensions of the grid
	// const unsigned gridWidth = gridDim.x * blockDim.x;
	// const unsigned gridHeight = gridDim.y * blockDim.y;

	// indices of each thread
	const int gridXidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gridYidx = blockIdx.y * blockDim.y + threadIdx.y;

	// matrix size information
	const int matrixXradius = filterMatrixWidth / 2;
	const int matrixYradius = filterMatrixHeight / 2;

	// shared memory allocation
	extern __shared__ byte smem[];
	byte* sharedPixelData = smem;
	float* sharedMatrixData = reinterpret_cast<float*>(sharedPixelData + (matrixXradius + (matrixXradius % 2) + blockDim.x) * (2 * matrixYradius + blockDim.y) * 8);

	// index calculations for loading global -> shared memory
	const int globalLoadXstart = gridXidx - (matrixXradius / 2 + matrixXradius % 2);
	const int globalLoadXreads = blockDim.x + 2 * (matrixXradius / 2 + matrixXradius % 2); // intended total # of reads by all threads combined
	const int globalLoadYreads = blockDim.y + 2 * matrixYradius;
	const int globalLoadXiterations = globalLoadXreads / blockDim.x + ((globalLoadXreads % blockDim.x) ? 1 : 0);
	const int globalLoadYiterations = globalLoadYreads / blockDim.y + ((globalLoadYreads % blockDim.y) ? 1 : 0);

	const int sharedStorePitch = (blockDim.x + matrixXradius + (matrixXradius % 2)) * 8;

	int globalLoadXindex = globalLoadXstart;
	int globalLoadYindex = gridYidx - matrixYradius;

	int sharedStoreXindex = threadIdx.x;
	int sharedStoreYindex = threadIdx.y;


	// perform loads
	for(int yStrides = 0; yStrides < globalLoadYiterations; ++yStrides)
	{
		if(sharedStoreYindex < globalLoadYreads)
		{
			for(int xStrides = 0; xStrides < globalLoadXiterations; ++xStrides)
			{
				if(sharedStoreXindex < globalLoadXreads)
				{
					reinterpret_cast<word*>(sharedPixelData + sharedStoreYindex * sharedStorePitch)[sharedStoreXindex]
					=
					reinterpret_cast<const word*>(static_cast<const byte*>(input) + min(max(globalLoadYindex, 0), pixelsHeight - 1) * pitchInput)[min(max(globalLoadXindex, 0), pixelsWidth / 2 - 1)];
				}

				globalLoadXindex += blockDim.x;
				sharedStoreXindex += blockDim.x;
			}

			globalLoadXindex = globalLoadXstart;
			sharedStoreXindex = threadIdx.x;
		}

		globalLoadYindex += blockDim.y;
		sharedStoreYindex += blockDim.y;
	}

	const unsigned blockSize1D = blockDim.x * blockDim.y;
	const unsigned blockIdx1D = blockDim.x * threadIdx.y + threadIdx.x;
	const unsigned filterMatrixSize1D = filterMatrixWidth * filterMatrixHeight;
	const unsigned moveIdx = filterMatrixSize1D + blockIdx1D - blockSize1D;
	if(blockIdx1D >= blockSize1D - filterMatrixSize1D)
	{
		sharedMatrixData[moveIdx] = filterMatrix[moveIdx];
	}

	// make sure all data is in shared memory after loads are done and before reads begin
	__syncthreads();

	// kill threads that are out of bounds
	if(gridXidx * 2 >= pixelsWidth || gridYidx >= pixelsHeight)
		return;

	word outputPair = 0xffffffffffffffff;

	byte inputValue = 0;
	float outputValue = 0;

	#pragma unroll
	for(int pixelInPair = 0; pixelInPair < 2; ++pixelInPair)
	{
		#pragma unroll
		for(int colorElement = 0; colorElement < 3; ++colorElement)
		{
			outputValue = 0;
			#pragma unroll
			for(int neighborYoffset = 0; neighborYoffset < filterMatrixHeight; ++neighborYoffset)
			{
				#pragma unroll
				for(int neighborXoffset = 0; neighborXoffset < filterMatrixWidth; ++neighborXoffset)
				{
					inputValue = sharedPixelData[(threadIdx.y + neighborYoffset) * sharedStorePitch + (2 * threadIdx.x + pixelInPair + neighborXoffset + matrixXradius % 2) * 4 + colorElement];
					outputValue += inputValue * sharedMatrixData[neighborYoffset * filterMatrixWidth + neighborXoffset];
				}
			}

			reinterpret_cast<sbyte*>(&outputPair)[pixelInPair * 4 + colorElement] = max(min(static_cast<int>(outputValue * 0.5f + 0.5f), 127), -128);
		}
	}

	reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput)[gridXidx] = outputPair;
}

// arrange read and write such that the one memory location can be used
// kernel must be given normalized convolution matrices of odd width and height
template<const int filterMatrixWidth, const int filterMatrixHeight>
__global__
void kernelMatrixConvolution32bit(const void* const input, const int pitchInput,
							 void* const output, const int pitchOutput,
							 const int pixelsWidth, const int pixelsHeight,
							 const float* const filterMatrix)
{
	// indices of each thread
	const int gridXidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gridYidx = blockIdx.y * blockDim.y + threadIdx.y;

	// matrix size information
	const int matrixXradius = filterMatrixWidth / 2;
	const int matrixYradius = filterMatrixHeight / 2;

	// shared memory allocation
	extern __shared__ byte smem[];
	byte* sharedPixelData = smem;
	float* sharedMatrixData = reinterpret_cast<float*>(sharedPixelData + (matrixXradius + (matrixXradius % 2) + blockDim.x) * (2 * matrixYradius + blockDim.y) * 8);

	// index calculations for loading global -> shared memory
	const int globalLoadXstart = gridXidx - (matrixXradius / 2 + matrixXradius % 2);
	const int globalLoadXreads = blockDim.x + 2 * (matrixXradius / 2 + matrixXradius % 2); // intended total # of reads by all threads combined
	const int globalLoadYreads = blockDim.y + 2 * matrixYradius;
	const int globalLoadXiterations = globalLoadXreads / blockDim.x + ((globalLoadXreads % blockDim.x) ? 1 : 0);
	const int globalLoadYiterations = globalLoadYreads / blockDim.y + ((globalLoadYreads % blockDim.y) ? 1 : 0);

	const int sharedStorePitch = (blockDim.x + matrixXradius + (matrixXradius % 2)) * 4;

	int globalLoadXindex = globalLoadXstart;
	int globalLoadYindex = gridYidx - matrixYradius;

	int sharedStoreXindex = threadIdx.x;
	int sharedStoreYindex = threadIdx.y;

	word inputPixelPair = 0;

	// perform loads
	for(int yStrides = 0; yStrides < globalLoadYiterations; ++yStrides)
	{
		if(sharedStoreYindex < globalLoadYreads)
		{
			for(int xStrides = 0; xStrides < globalLoadXiterations; ++xStrides)
			{
				if(sharedStoreXindex < globalLoadXreads)
				{
					// // get pixel pair
					// reinterpret_cast<word*>(sharedPixelData + sharedStoreYindex * sharedStorePitch)[sharedStoreXindex]
					inputPixelPair
					=
					reinterpret_cast<const word*>(static_cast<const byte*>(input) + min(max(globalLoadYindex, 0), pixelsHeight - 1) * pitchInput)[min(max(globalLoadXindex, 0), pixelsWidth / 2 - 1)];

					// write to two separate surfaces in shared memory
					reinterpret_cast<uint32_t*>(sharedPixelData + sharedStoreYindex * sharedStorePitch)[sharedStoreXindex] = reinterpret_cast<uint32_t*>(&inputPixelPair)[0];
					reinterpret_cast<uint32_t*>(sharedPixelData + (sharedStoreYindex + globalLoadYreads) * sharedStorePitch)[sharedStoreXindex] = reinterpret_cast<uint32_t*>(&inputPixelPair)[1];
				}

				globalLoadXindex += blockDim.x;
				sharedStoreXindex += blockDim.x;
			}

			globalLoadXindex = globalLoadXstart;
			sharedStoreXindex = threadIdx.x;
		}

		globalLoadYindex += blockDim.y;
		sharedStoreYindex += blockDim.y;
	}

	const unsigned blockSize1D = blockDim.x * blockDim.y;
	const unsigned blockIdx1D = blockDim.x * threadIdx.y + threadIdx.x;
	const unsigned filterMatrixSize1D = filterMatrixWidth * filterMatrixHeight;
	const unsigned moveIdx = filterMatrixSize1D + blockIdx1D - blockSize1D;
	if(blockIdx1D >= blockSize1D - filterMatrixSize1D)
	{
		sharedMatrixData[moveIdx] = filterMatrix[moveIdx];
	}

	// make sure all data is in shared memory after loads are done and before reads begin
	__syncthreads();

	// kill threads that are out of bounds
	if(gridXidx * 2 >= pixelsWidth || gridYidx >= pixelsHeight)
		return;

	word outputPair = 0xffffffffffffffff;

	byte inputValue = 0;
	float outputValue = 0;

	#pragma unroll
	for(int pixelInPair = 0; pixelInPair < 2; ++pixelInPair)
	{
		#pragma unroll
		for(int colorElement = 0; colorElement < 3; ++colorElement)
		{
			outputValue = 0;
			#pragma unroll
			for(int neighborYoffset = 0; neighborYoffset < filterMatrixHeight; ++neighborYoffset)
			{
				#pragma unroll
				for(int neighborXoffset = 0; neighborXoffset < filterMatrixWidth; ++neighborXoffset)
				{
					inputValue = sharedPixelData[(threadIdx.y + neighborYoffset + pixelInPair * globalLoadYreads) * sharedStorePitch + (threadIdx.x + neighborXoffset + matrixXradius % 2) * 4 + colorElement];
					outputValue += inputValue * sharedMatrixData[neighborYoffset * filterMatrixWidth + neighborXoffset];
				}
			}

			reinterpret_cast<sbyte*>(&outputPair)[pixelInPair * 4 + colorElement] = max(min(static_cast<int>(outputValue * 0.5f + 0.5f), 127), -128);
		}
	}

	reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput)[gridXidx] = outputPair;
}

// magnitude of sum of A and B treated as orthogonal axis-aligned vectors
// assumes input as an array of signed 8-bit integer values
#define VEC_SUM_GRID_STRIDE_COUNT 8 // number of sections the image is processed in
__global__
void kernelVectorSum(const void* const inputA, const unsigned pitchInputA,
					 const void* const inputB, const unsigned pitchInputB,
					 void* const output, const unsigned pitchOutput,
					 const unsigned pixelsWidth, const unsigned pixelsHeight)
{
	// const unsigned gridWidth = gridDim.x * blockDim.x;
	const unsigned activeGridWidth = pixelsWidth / (2 * VEC_SUM_GRID_STRIDE_COUNT);

	// indices of each thread
	const unsigned gridXidx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned gridYidx = blockIdx.y * blockDim.y + threadIdx.y;

	// kill threads that are out of bounds
	if(gridXidx >= activeGridWidth || gridYidx >= pixelsHeight)
		return;

	// destinations for packed data
	word inputApixels = 0, inputBpixels = 0, outputCpixels = 0;

	// work on the image in multiple parts with fewer threads
	#pragma unroll
	for(int i = 0; i < VEC_SUM_GRID_STRIDE_COUNT; ++i)
	{
		// read
		inputApixels = reinterpret_cast<const word*>(static_cast<const byte*>(inputA) + gridYidx * pitchInputA)[i * activeGridWidth + gridXidx];
		inputBpixels = reinterpret_cast<const word*>(static_cast<const byte*>(inputB) + gridYidx * pitchInputB)[i * activeGridWidth + gridXidx];

		// store the vector magnitude of each component
		reinterpret_cast<byte*>(&outputCpixels)[0] = min(2 * sqrt(static_cast<float>(reinterpret_cast<sbyte*>(&inputApixels)[0]) * reinterpret_cast<sbyte*>(&inputApixels)[0] + static_cast<float>(reinterpret_cast<sbyte*>(&inputBpixels)[0]) * reinterpret_cast<sbyte*>(&inputBpixels)[0]) + 0.5f, 255.f);
		reinterpret_cast<byte*>(&outputCpixels)[1] = min(2 * sqrt(static_cast<float>(reinterpret_cast<sbyte*>(&inputApixels)[1]) * reinterpret_cast<sbyte*>(&inputApixels)[1] + static_cast<float>(reinterpret_cast<sbyte*>(&inputBpixels)[1]) * reinterpret_cast<sbyte*>(&inputBpixels)[1]) + 0.5f, 255.f);
		reinterpret_cast<byte*>(&outputCpixels)[2] = min(2 * sqrt(static_cast<float>(reinterpret_cast<sbyte*>(&inputApixels)[2]) * reinterpret_cast<sbyte*>(&inputApixels)[2] + static_cast<float>(reinterpret_cast<sbyte*>(&inputBpixels)[2]) * reinterpret_cast<sbyte*>(&inputBpixels)[2]) + 0.5f, 255.f);

		reinterpret_cast<byte*>(&outputCpixels)[4] = min(2 * sqrt(static_cast<float>(reinterpret_cast<sbyte*>(&inputApixels)[4]) * reinterpret_cast<sbyte*>(&inputApixels)[4] + static_cast<float>(reinterpret_cast<sbyte*>(&inputBpixels)[4]) * reinterpret_cast<sbyte*>(&inputBpixels)[4]) + 0.5f, 255.f);
		reinterpret_cast<byte*>(&outputCpixels)[5] = min(2 * sqrt(static_cast<float>(reinterpret_cast<sbyte*>(&inputApixels)[5]) * reinterpret_cast<sbyte*>(&inputApixels)[5] + static_cast<float>(reinterpret_cast<sbyte*>(&inputBpixels)[5]) * reinterpret_cast<sbyte*>(&inputBpixels)[5]) + 0.5f, 255.f);
		reinterpret_cast<byte*>(&outputCpixels)[6] = min(2 * sqrt(static_cast<float>(reinterpret_cast<sbyte*>(&inputApixels)[6]) * reinterpret_cast<sbyte*>(&inputApixels)[6] + static_cast<float>(reinterpret_cast<sbyte*>(&inputBpixels)[6]) * reinterpret_cast<sbyte*>(&inputBpixels)[6]) + 0.5f, 255.f);

		// write
		reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput)[i * activeGridWidth + gridXidx] = outputCpixels;
	}
}

// matrix sum of -A and B
#define DIFFERENCE_GRID_STRIDE_COUNT 8 // number of sections the image is processed in
__global__
void kernelMatrixDifference(const void* const inputA, const unsigned pitchInputA,
						    const void* const inputB, const unsigned pitchInputB,
						    void* const output, const unsigned pitchOutput,
						    const unsigned pixelsWidth, const unsigned pixelsHeight)
{
	// const unsigned gridWidth = gridDim.x * blockDim.x;
	const unsigned activeGridWidth = pixelsWidth / (2 * DIFFERENCE_GRID_STRIDE_COUNT);

	// indices of each thread
	const unsigned gridXidx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned gridYidx = blockIdx.y * blockDim.y + threadIdx.y;

	// kill threads that are out of bounds
	if(gridXidx >= activeGridWidth || gridYidx >= pixelsHeight)
		return;

	// destinations for packed data
	word inputApixels = 0, inputBpixels = 0, outputCpixels = 0;

	// work on the image in multiple parts with fewer threads
	#pragma unroll
	for(int i = 0; i < DIFFERENCE_GRID_STRIDE_COUNT; ++i)
	{
		// read
		inputApixels = reinterpret_cast<const word*>(static_cast<const byte*>(inputA) + gridYidx * pitchInputA)[i * activeGridWidth + gridXidx];
		inputBpixels = reinterpret_cast<const word*>(static_cast<const byte*>(inputB) + gridYidx * pitchInputB)[i * activeGridWidth + gridXidx];

		// store the difference of each component
		reinterpret_cast<byte*>(&outputCpixels)[0] = max(min(static_cast<int>(reinterpret_cast<byte*>(&inputBpixels)[0]) - reinterpret_cast<byte*>(&inputApixels)[0], 255), 0);
		reinterpret_cast<byte*>(&outputCpixels)[1] = max(min(static_cast<int>(reinterpret_cast<byte*>(&inputBpixels)[1]) - reinterpret_cast<byte*>(&inputApixels)[1], 255), 0);
		reinterpret_cast<byte*>(&outputCpixels)[2] = max(min(static_cast<int>(reinterpret_cast<byte*>(&inputBpixels)[2]) - reinterpret_cast<byte*>(&inputApixels)[2], 255), 0);

		reinterpret_cast<byte*>(&outputCpixels)[4] = max(min(static_cast<int>(reinterpret_cast<byte*>(&inputBpixels)[4]) - reinterpret_cast<byte*>(&inputApixels)[4], 255), 0);
		reinterpret_cast<byte*>(&outputCpixels)[5] = max(min(static_cast<int>(reinterpret_cast<byte*>(&inputBpixels)[5]) - reinterpret_cast<byte*>(&inputApixels)[5], 255), 0);
		reinterpret_cast<byte*>(&outputCpixels)[6] = max(min(static_cast<int>(reinterpret_cast<byte*>(&inputBpixels)[6]) - reinterpret_cast<byte*>(&inputApixels)[6], 255), 0);

		// write
		reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput)[i * activeGridWidth + gridXidx] = outputCpixels;
	}
}

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 8

// (maybe) when this works, modify it to push to a ConcurrentQueue<GPUFrame>
// allocate new space before converting
GPUFrame NV12toRGBA(GPUFrame& NV12input)
{
	// reference for the new frame
	GPUFrame allocatedFrame;

	// make an object for the output image
	unsigned allocationRows = NV12input.height();
	unsigned allocationCols = 4 * NV12input.width();

	// make the actual memory allocation
	allocatedFrame = GPUFrame(NV12input.width(), NV12input.height(), allocationCols, allocationRows, NV12input.timestamp());

	if(0 == NV12toRGBA(NV12input, allocatedFrame))
	{
		// original success indicator
		return allocatedFrame;
	}
	else
	{
		// original failure indicator
		return GPUFrame();
	}
}

// run conversion kernel with pre-allocated output memory
// return 0 on success, anything else on failure
// TODO: switch statement for common sizes and template call for ones needing padding
int NV12toRGBA(GPUFrame& NV12input, GPUFrame& RGBAoutput)
{
	// make sure the width and height divide nicely
	bool matchedWidth = !(NV12input.width() % (8 * BLOCK_WIDTH));
	bool matchedHeight = !(NV12input.height() % BLOCK_HEIGHT);

	if(matchedWidth && matchedHeight)
	{
		// dimensions for kernel launch
		dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
		dim3 grid(NV12input.width() / (8 * block.x), NV12input.height() / block.y);

		kernelNV12toRGBA<false><<< grid, block >>>(NV12input.data(), NV12input.pitch(),
												   RGBAoutput.data(), RGBAoutput.pitch());

		// sync and check for errors
		hipDeviceSynchronize(); cudaErr(hipGetLastError());
	}
	else
	{
		// dimensions for kernel launch
		dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
		dim3 grid(NV12input.width() / (8 * block.x), NV12input.height() / block.y);

		// add in a block of width and/or height to reach all pixels
		if(!matchedWidth)
			grid.x++;

		if(!matchedHeight)
			grid.y++;

		kernelNV12toRGBA<true><<< grid, block >>>(NV12input.data(), NV12input.pitch(),
												  RGBAoutput.data(), RGBAoutput.pitch(),
												  RGBAoutput.width(), RGBAoutput.height());

		// sync and check for errors
		hipDeviceSynchronize(); cudaErr(hipGetLastError());
	}

	return 0; // success
}

// (maybe) when this works, modify it to push to a ConcurrentQueue<GPUFrame>
// allocate new space before converting
GPUFrame NV12toGrayscale(GPUFrame& NV12input)
{
	// reference for the new frame
	GPUFrame allocatedFrame;

	// make an object for the output image
	unsigned allocationRows = NV12input.height();
	unsigned allocationCols = 4 * NV12input.width();

	// make the actual memory allocation
	allocatedFrame = GPUFrame(NV12input.width(), NV12input.height(), allocationCols, allocationRows, NV12input.timestamp());

	if(0 == NV12toGrayscale(NV12input, allocatedFrame))
	{
		// original success indicator
		return allocatedFrame;
	}
	else
	{
		// original failure indicator
		return GPUFrame();
	}
}

// run conversion kernel with pre-allocated output memory
// return 0 on success, anything else on failure
// TODO: switch statement for common sizes and template call for ones needing padding
int NV12toGrayscale(GPUFrame& NV12input, GPUFrame& grayOutput)
{
	// dimensions for kernel launch
	dim3 grid, block(BLOCK_WIDTH, BLOCK_HEIGHT);
	grid.x = NV12input.width() / (8 * block.x);
	grid.y = NV12input.height() / block.y;

	// add in a block of width and/or height to reach all pixels
	if(NV12input.width() % (8 * block.x))
		grid.x++;

	if(NV12input.height() % block.y)
		grid.y++;


	kernelNV12toMAG<<< grid, block >>>(NV12input.data(), NV12input.pitch(),
									   grayOutput.data(), grayOutput.pitch(),
									   grayOutput.width(), grayOutput.height());

	// sync and check for errors
	hipDeviceSynchronize(); cudaErr(hipGetLastError());

	// success
	return 0;
}

GPUFrame RGBAtoGrayscale(GPUFrame& RGBAinput)
{
	// reference for the new frame
	GPUFrame allocatedFrame;

	// make an object for the output image
	unsigned allocationRows = RGBAinput.height();
	unsigned allocationCols = 4 * RGBAinput.width();

	// make the actual memory allocation
	allocatedFrame = GPUFrame(RGBAinput.width(), RGBAinput.height(), allocationCols, allocationRows, RGBAinput.timestamp());

	if(0 == RGBAtoGrayscale(RGBAinput, allocatedFrame))
	{
		// original success indicator
		return allocatedFrame;
	}
	else
	{
		// original failure indicator
		return GPUFrame();
	}
}

int RGBAtoGrayscale(GPUFrame& RGBAinput, GPUFrame& grayOutput)
{
	// dimension the grid
	dim3 grid, block(BLOCK_WIDTH, BLOCK_HEIGHT);
	grid.x = RGBAinput.width() / (2 * RGBA_MAG_GRID_STRIDE_COUNT * block.x);
	grid.y = RGBAinput.height() / block.y;

	if(RGBAinput.width() % (2 * RGBA_MAG_GRID_STRIDE_COUNT * block.x))
		grid.x++;

	if(RGBAinput.height() % block.y)
		grid.y++;

	kernelRGBAtoMAG<<< grid, block >>>(RGBAinput.data(), RGBAinput.pitch(),
									   grayOutput.data(), grayOutput.pitch(),
									   RGBAinput.width(), RGBAinput.height());

	// sync and check for errors
	hipDeviceSynchronize(); cudaErr(hipGetLastError());

	// success
	return 0;
}

// allocate for and run the sobel filter
GPUFrame sobelFilter(GPUFrame& image)
{
	// reference for the new frame
	GPUFrame allocatedFrame;

	// make an object for the output image
	unsigned allocationRows = image.height();
	unsigned allocationCols = 4 * image.width();

	// make the actual memory allocation
	allocatedFrame = GPUFrame(image.width(), image.height(), allocationCols, allocationRows, image.timestamp());

	if(0 == sobelFilter(image, allocatedFrame))
	{
		// original success indicator
		return allocatedFrame;
	}
	else
	{
		// original failure indicator
		return GPUFrame();
	}
}

// launch sobel filter kernel
int sobelFilter(GPUFrame& image, GPUFrame& edges)
{
	// keep static device pointer to normalized sobel
	// convolution filter and generate if first call

	static float hostSobelXFilter[] = {-1.f/16, 0.f, 1.f/16, -2.f/16, 0.f, 2.f/16, -1.f/16, 0.f, 1.f/16};
	// static float hostSobelXFilter[] = {0.f, 0.f, 0.f, 0.f, 1.f, 0.f, 0.f, 0.f, 0.f}; // identity matrix for testing
	static float* sobelXFilter = nullptr;
	static float hostSobelYFilter[] = {-1.f/16, -2.f/16, -1.f/16, 0, 0, 0, 1.f/16, 2.f/16, 1.f/16};
	// static float hostSobelYFilter[] = {0.f, 0.f, 0.f, 0.f, 1.f, 0.f, 0.f, 0.f, 0.f}; // identity matrix for testing
	static float* sobelYFilter = nullptr;

	// space for intermediate results
	static GPUFrame sobelX, sobelY;

	// one-time host-to-device copy of the filters
	// one-time allocation of scratch surfaces
	if(nullptr == sobelXFilter)
	{
		// copy to device
		cudaErr(hipMalloc(&sobelXFilter, 9 * sizeof(float)));
		cudaErr(hipMemcpy(sobelXFilter, hostSobelXFilter, 9 * sizeof(float), hipMemcpyHostToDevice));

		cudaErr(hipMalloc(&sobelYFilter, 9 * sizeof(float)));
		cudaErr(hipMemcpy(sobelYFilter, hostSobelYFilter, 9 * sizeof(float), hipMemcpyHostToDevice));

		// allocate frames for intermediate results
		sobelX = GPUFrame(image.width(), image.height(), 4 * image.width(), image.height(), 0);
		sobelY = GPUFrame(image.width(), image.height(), 4 * image.width(), image.height(), 0);
	}

	// figure out dimensions
	dim3 grid, block(BLOCK_WIDTH, BLOCK_HEIGHT);
	grid.x = image.width() / (2 * block.x);
	grid.y = image.height() / block.y;

	if(image.width() % (2 * block.x))
		grid.x++;

	if(image.height() % block.y)
		grid.y++;

	// shared memory size
	unsigned sharedSpaceSize = 0;
	sharedSpaceSize += (4 + 2 * block.x) * (2 + block.y) * 4; // pixel data
	sharedSpaceSize += 3 * 3 * sizeof(float); // convolution matrix

	// launch convolution kernel with sobel matrix
	kernelMatrixConvolution<3, 3><<< grid, block, sharedSpaceSize >>>(image.data(), image.pitch(),
																sobelX.data(), sobelX.pitch(),
																image.width(), image.height(),
																sobelXFilter);

	kernelMatrixConvolution<3, 3><<< grid, block, sharedSpaceSize >>>(image.data(), image.pitch(),
																sobelY.data(), sobelY.pitch(),
																image.width(), image.height(),
																sobelYFilter);

	// 	// launch convolution kernel with sobel matrix
	// kernelMatrixConvolution32bit<3, 3><<< grid, block, sharedSpaceSize >>>(image.data(), image.pitch(),
	// 															sobelX.data(), sobelX.pitch(),
	// 															image.width(), image.height(),
	// 															sobelXFilter);

	// kernelMatrixConvolution32bit<3, 3><<< grid, block, sharedSpaceSize >>>(image.data(), image.pitch(),
	// 															sobelY.data(), sobelY.pitch(),
	// 															image.width(), image.height(),
	// 															sobelYFilter);
	// 	// launch convolution kernel with sobel matrix
	// kernelMatrixConvolution32bit<3, 3><<< grid, block, sharedSpaceSize >>>(image.data(), image.pitch(),
	// 															edges.data(), edges.pitch(),
	// 															image.width(), image.height(),
	// 															sobelXFilter);

	// width of the grid must change for vector sum
	grid.x = image.width() / (2 * VEC_SUM_GRID_STRIDE_COUNT * block.x);
	if(0 != image.width() % (2 * VEC_SUM_GRID_STRIDE_COUNT * block.x))
		grid.x++;

	// vector sum of both sobel images
	kernelVectorSum<<< grid, block >>>(sobelX.data(), sobelX.pitch(),
									   sobelY.data(), sobelY.pitch(),
									   edges.data(), edges.pitch(),
									   image.width(), image.height());

	// sync and check for errors
	hipDeviceSynchronize(); cudaErr(hipGetLastError());

	// success
	return 0;
}

// allocate for and run the difference kernel
GPUFrame matrixDifference(GPUFrame& positive, GPUFrame& negative)
{
	// reference for the new frame
	GPUFrame allocatedFrame;

	// make an object for the output image
	unsigned allocationRows = positive.height();
	unsigned allocationCols = 4 * positive.width();

	// make the actual memory allocation
	allocatedFrame = GPUFrame(positive.width(), positive.height(), allocationCols, allocationRows, positive.timestamp());

	if(0 == matrixDifference(positive, negative, allocatedFrame))
	{
		// original success indicator
		return allocatedFrame;
	}
	else
	{
		// original failure indicator
		return GPUFrame();
	}
}

int matrixDifference(GPUFrame& positive, GPUFrame& negative, GPUFrame& difference)
{
	// figure out dimensions
	dim3 grid, block(BLOCK_WIDTH, BLOCK_HEIGHT);
	grid.x = positive.width() / (2 * block.x);
	grid.y = positive.height() / block.y;

	if(positive.width() % (2 * block.x))
		grid.x++;

	if(positive.height() % block.y)
		grid.y++;

	// vector sum of both sobel images
	kernelMatrixDifference<<< grid, block >>>(negative.data(), negative.pitch(),
											  positive.data(), positive.pitch(),
											  difference.data(), difference.pitch(),
											  positive.width(), positive.height());

	// sync and check for errors
	hipDeviceSynchronize(); cudaErr(hipGetLastError());

	// success
	return 0;
}