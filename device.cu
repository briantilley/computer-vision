#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "headers/device.h"

// development
#include <iostream>

// #define YUV_TO_RGB_FLOAT

// convert to RGB using ITU 601 standard
__device__ // function for GPU thread to use
inline void YUVtoRGBA(const byte& Y, const byte& U, const byte& V, byte& R, byte& G, byte& B, byte& A)
{
	#ifdef YUV_TO_RGB_FLOAT
		float fY, fU, fV;

		fY = Y * 298.082f / 256.f;
		fU = U / 256.f;
		fV = V / 256.f;

		R = min(max(static_cast<int>(fY + 408.593f * fV - 222.921), 0), 255);
		G = min(max(static_cast<int>(fY - 100.291f * fU - 208.12f * fV + 135.576), 0), 255);
		B = min(max(static_cast<int>(fY + 516.412 * fU - 276.836), 0), 255);
	#else
		int tY = 298 * Y;

		R = min(max((tY + 409 * V - 57068) >> 8, 0), 255);
		G = min(max((tY - 100 * U - 208 * V + 34707 >> 8), 0), 255);
		B = min(max((tY + 516 * U - 70870) >> 8, 0), 255);
	#endif

	// no alpha data in YUV
	A = 255;
}

// set 'clampCoords' to false when grid size
// perfectly corresponds to image size
// (each thread is 8 pixels wide)
template<bool clampCoords=false>
__global__
void kernelNV12toRGBA(const void* const input, const unsigned pitchInput,
					  void* const output, const unsigned pitchOutput,
					  const unsigned pixelsWidth=0, const unsigned pixelsHeight=0)
{
	// make sure we get the right data for clamping coords if necessary
	if(clampCoords)
	{
		if(!(pixelsWidth && pixelsHeight))
			return;
	}

	// // number of rows of threads in use
	// const unsigned gridWidth = gridDim.x * blockDim.x;
	unsigned gridHeight;
	if(clampCoords)
		gridHeight = pixelsHeight;
	else
		gridHeight = gridDim.y * blockDim.y;

	// position within the grid
	// (threads per block) * (position of block in grid) + (position of thread in block)
	const unsigned gridXidx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned gridYidx = blockDim.y * blockIdx.y + threadIdx.y;

	if(clampCoords)
	{
		if(gridXidx * 8 >= pixelsWidth || gridYidx >= pixelsHeight)
			return;
	}

	// NV12 global reads for 8 pixels of data
	// address calculation from inner to outer access:
	// convert to byte array, position to proper row,
	// convert to row array, position to proper column
	word packed_Y_bytes, packed_UV_bytes;
	packed_Y_bytes = reinterpret_cast<const word*>(static_cast<const byte*>(input) + gridYidx * pitchInput)[gridXidx];
	packed_UV_bytes = reinterpret_cast<const word*>(static_cast<const byte*>(input) + (gridHeight + gridYidx / 2) * pitchInput)[gridXidx];

	// array representation of input data
	const byte* const Y  = reinterpret_cast<const byte*>(&packed_Y_bytes);
	const byte* const UV = reinterpret_cast<const byte*>(&packed_UV_bytes);

	// local destination for conversion
	word pixelPairs[4]; // pack into 4 2-pixel/8-byte pairs
	byte* const pair_0_bytes = reinterpret_cast<byte*>(pixelPairs    );
	byte* const pair_1_bytes = reinterpret_cast<byte*>(pixelPairs + 1);
	byte* const pair_2_bytes = reinterpret_cast<byte*>(pixelPairs + 2);
	byte* const pair_3_bytes = reinterpret_cast<byte*>(pixelPairs + 3);

	// this method of computation exposes ILP
	// convert all of the pixel data
	YUVtoRGBA(Y[0], UV[0], UV[1], pair_0_bytes[0], pair_0_bytes[1], pair_0_bytes[2], pair_0_bytes[3]);
	YUVtoRGBA(Y[1], UV[0], UV[1], pair_0_bytes[4], pair_0_bytes[5], pair_0_bytes[6], pair_0_bytes[7]);
	YUVtoRGBA(Y[2], UV[2], UV[3], pair_1_bytes[0], pair_1_bytes[1], pair_1_bytes[2], pair_1_bytes[3]);
	YUVtoRGBA(Y[3], UV[2], UV[3], pair_1_bytes[4], pair_1_bytes[5], pair_1_bytes[6], pair_1_bytes[7]);
	YUVtoRGBA(Y[4], UV[4], UV[5], pair_2_bytes[0], pair_2_bytes[1], pair_2_bytes[2], pair_2_bytes[3]);
	YUVtoRGBA(Y[5], UV[4], UV[5], pair_2_bytes[4], pair_2_bytes[5], pair_2_bytes[6], pair_2_bytes[7]);
	YUVtoRGBA(Y[6], UV[6], UV[7], pair_3_bytes[0], pair_3_bytes[1], pair_3_bytes[2], pair_3_bytes[3]);
	YUVtoRGBA(Y[7], UV[6], UV[7], pair_3_bytes[4], pair_3_bytes[5], pair_3_bytes[6], pair_3_bytes[7]);

	// strided global write of the RGBA data for 8 pixels,
	// taking the hit on efficiency
	word* const row = reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput);
	const unsigned firstColumn = 4 * gridXidx; 

	row[firstColumn    ] = pixelPairs[0];
	row[firstColumn + 1] = pixelPairs[1];
	row[firstColumn + 2] = pixelPairs[2];
	row[firstColumn + 3] = pixelPairs[3];
}

// wil be 8 pixels per thread, load 2 at a time with grid-strided reads
// arrange read and write such that the one memory location can be used
// kernel must be given normalized convolution matrices of odd width and height
__global__
void kernelMatrixConvolution(const void* const input, const unsigned pitchInput,
							 void* const output, const unsigned pitchOutput,
							 const unsigned pixelsWidth, const unsigned pixelsHeight,
							 const float* const filterMatrix,
							 const unsigned filterMatrixWidth, const unsigned filterMatrixHeight)
{
	// // dimensions of the grid
	// const unsigned gridWidth = gridDim.x * blockDim.x;
	// const unsigned gridHeight = gridDim.y * blockDim.y;

	// indices of each thread
	const unsigned gridXidx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned gridYidx = blockIdx.y * blockDim.y + threadIdx.y;

	// kill threads that are out of bounds
	if(gridXidx >= pixelsWidth || gridYidx >= pixelsHeight)
		return;

	int newVal = 0; // new RGBA values for the two pixels
	float readVal = 0; // two pixels read from input

	const int firstRow = gridYidx - filterMatrixHeight / 2;
	const int firstColumn = gridXidx - filterMatrixWidth / 2;

	for(int colorElement = 0; colorElement < 3; ++colorElement)
	{
		newVal = 0;

		for(int i = 0; i < filterMatrixHeight; ++i)
		{
			for(int j = 0; j < filterMatrixWidth; ++j)
			{
				readVal = static_cast<const byte*>(input)[min(max(firstRow + i, 0), pixelsHeight - 1) * pitchInput + min(max(firstColumn + j, 0), pixelsWidth - 1) * 4 + colorElement];
				newVal += readVal * filterMatrix[i * filterMatrixWidth + j] + 0.5f;
			}
		}

		static_cast<byte*>(output)[gridYidx * pitchOutput + gridXidx * 4 + colorElement] = min(min(newVal, 0), 255);
	}
}

// sum of magnitudes of A and B treated as legs of a right triangle
// should be 2 pixels per thread
__global__
void kernelVectorSum(const void* const inputA, const unsigned pitchInputA,
					 const void* const inputB, const unsigned pitchInputB,
					 void* const output, const unsigned pitchOutput,
					 const unsigned pixelsWidth, const unsigned pixelsHeight)
{
	const unsigned gridWidth = gridDim.x * blockDim.x;

	// indices of each thread
	const unsigned gridXidx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned gridYidx = blockIdx.y * blockDim.y + threadIdx.y;

	// kill threads that are out of bounds
	if(/*gridXidx * 2 >= pixelsWidth || */gridYidx >= pixelsHeight)
		return;

	// destinations for packed data
	word inputApixels = 0, inputBpixels = 0, outputCpixels = 0;

	// read
	inputApixels = reinterpret_cast<const word*>(static_cast<const byte*>(inputA) + gridYidx * pitchInputA)[gridXidx];
	inputBpixels = reinterpret_cast<const word*>(static_cast<const byte*>(inputB) + gridYidx * pitchInputB)[gridXidx];

	if(gridXidx >= gridWidth / 2)
	{
		// inputApixels = 0xffffffffffffffff;
		// inputBpixels = 0xffffffffffffffff;
	}

	// store the vector magnitude of each component
	reinterpret_cast<byte*>(&outputCpixels)[0] = sqrt(static_cast<float>(reinterpret_cast<const byte*>(&inputApixels)[0]) * reinterpret_cast<const byte*>(&inputApixels)[0] + static_cast<float>(reinterpret_cast<const byte*>(&inputBpixels)[0]) * reinterpret_cast<const byte*>(&inputBpixels)[0]);
	reinterpret_cast<byte*>(&outputCpixels)[1] = sqrt(static_cast<float>(reinterpret_cast<const byte*>(&inputApixels)[1]) * reinterpret_cast<const byte*>(&inputApixels)[1] + static_cast<float>(reinterpret_cast<const byte*>(&inputBpixels)[1]) * reinterpret_cast<const byte*>(&inputBpixels)[1]);
	reinterpret_cast<byte*>(&outputCpixels)[2] = sqrt(static_cast<float>(reinterpret_cast<const byte*>(&inputApixels)[2]) * reinterpret_cast<const byte*>(&inputApixels)[2] + static_cast<float>(reinterpret_cast<const byte*>(&inputBpixels)[2]) * reinterpret_cast<const byte*>(&inputBpixels)[2]);

	reinterpret_cast<byte*>(&outputCpixels)[4] = sqrt(static_cast<float>(reinterpret_cast<const byte*>(&inputApixels)[4]) * reinterpret_cast<const byte*>(&inputApixels)[4] + static_cast<float>(reinterpret_cast<const byte*>(&inputBpixels)[4]) * reinterpret_cast<const byte*>(&inputBpixels)[4]);
	reinterpret_cast<byte*>(&outputCpixels)[5] = sqrt(static_cast<float>(reinterpret_cast<const byte*>(&inputApixels)[5]) * reinterpret_cast<const byte*>(&inputApixels)[5] + static_cast<float>(reinterpret_cast<const byte*>(&inputBpixels)[5]) * reinterpret_cast<const byte*>(&inputBpixels)[5]);
	reinterpret_cast<byte*>(&outputCpixels)[6] = sqrt(static_cast<float>(reinterpret_cast<const byte*>(&inputApixels)[6]) * reinterpret_cast<const byte*>(&inputApixels)[6] + static_cast<float>(reinterpret_cast<const byte*>(&inputBpixels)[6]) * reinterpret_cast<const byte*>(&inputBpixels)[6]);

	// write
	reinterpret_cast<word*>(static_cast<byte*>(output) + gridYidx * pitchOutput)[gridXidx] = outputCpixels;
}

// (maybe) when this works, modify it to push to a ConcurrentQueue<GPUFrame>
// allocate new space before converting
GPUFrame NV12toRGBA(GPUFrame& NV12input)
{
	// reference for the new frame
	GPUFrame allocatedFrame;

	// make an object for the output image
	unsigned allocationRows = NV12input.height();
	unsigned allocationCols = 4 * NV12input.width();

	// make the actual memory allocation
	allocatedFrame = GPUFrame(NV12input.width(), NV12input.height(), allocationCols, allocationRows, NV12input.timestamp());

	if(0 == NV12toRGBA(NV12input, allocatedFrame))
	{
		// original success indicator
		return allocatedFrame;
	}
	else
	{
		// original failure indicator
		return GPUFrame();
	}
}

// run conversion kernel with pre-allocated output memory
// return 0 on success, anything else on failure
// TODO: switch statement for common sizes and template call for ones needing padding
#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 8
int NV12toRGBA(GPUFrame& NV12input, GPUFrame& RGBAoutput)
{
	// make sure the width and height divide nicely
	bool matchedWidth = !(NV12input.width() % (8 * BLOCK_WIDTH));
	bool matchedHeight = !(NV12input.height() % BLOCK_HEIGHT);

	if(matchedWidth && matchedHeight)
	{
		// dimensions for kernel launch
		dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
		dim3 grid(NV12input.width() / (8 * block.x), NV12input.height() / block.y);

		kernelNV12toRGBA<false><<< grid, block >>>(NV12input.data(), NV12input.pitch(),
												   RGBAoutput.data(), RGBAoutput.pitch());

		// sync and check for errors
		hipDeviceSynchronize(); cudaErr(hipGetLastError());
	}
	else
	{
		// dimensions for kernel launch
		dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
		dim3 grid(NV12input.width() / (8 * block.x), NV12input.height() / block.y);

		// add in a block of width and/or height to reach all pixels
		if(!matchedWidth)
			grid.x++;

		if(!matchedHeight)
			grid.y++;

		kernelNV12toRGBA<true><<< grid, block >>>(NV12input.data(), NV12input.pitch(),
												  RGBAoutput.data(), RGBAoutput.pitch(),
												  RGBAoutput.width(), RGBAoutput.height());

		// sync and check for errors
		hipDeviceSynchronize(); cudaErr(hipGetLastError());
	}

	return 0; // success
}

// allocate for and run the sobel filter
GPUFrame sobelFilter(GPUFrame& image)
{
	// reference for the new frame
	GPUFrame allocatedFrame;

	// make an object for the output image
	unsigned allocationRows = image.height();
	unsigned allocationCols = 4 * image.width();

	// make the actual memory allocation
	allocatedFrame = GPUFrame(image.width(), image.height(), allocationCols, allocationRows, image.timestamp());

	if(0 == sobelFilter(image, allocatedFrame))
	{
		// original success indicator
		return allocatedFrame;
	}
	else
	{
		// original failure indicator
		return GPUFrame();
	}
}

// launch sobel filter kernel
int sobelFilter(GPUFrame& image, GPUFrame& edges)
{
	// keep static device pointer to normalized sobel
	// convolution filter and generate if first call
	static float hostSobelXFilter[] = {-1.f/8, 0.f, 1.f/8, -2.f/8, 0.f, 2.f/8, -1.f/8, 0.f, 1.f/8};
	static float* sobelXFilter = nullptr;
	static float hostSobelYFilter[] = {-1.f/8, -2.f/8, -1.f/8, 0, 0, 0, 1.f/8, 2.f/8, 1.f/8};
	static float* sobelYFilter = nullptr;

	if(nullptr == sobelXFilter)
	{
		cudaErr(hipMalloc(&sobelXFilter, 9 * sizeof(float)));
		cudaErr(hipMemcpy(sobelXFilter, hostSobelXFilter, 9 * sizeof(float), hipMemcpyHostToDevice));

		cudaErr(hipMalloc(&sobelYFilter, 9 * sizeof(float)));
		cudaErr(hipMemcpy(sobelYFilter, hostSobelYFilter, 9 * sizeof(float), hipMemcpyHostToDevice));
	}

	static GPUFrame sobelX(image.width(), image.height(), 4 * image.width(), image.height(), 0);
	static GPUFrame sobelY(image.width(), image.height(), 4 * image.width(), image.height(), 0);

	// figure out dimensions
	dim3 grid, block(BLOCK_WIDTH, BLOCK_HEIGHT);
	grid.x = image.width() / (1 * block.x);
	grid.y = image.height() / block.y;

	if(image.width() % (1 * block.x))
		grid.x++;

	if(image.height() % block.y)
		grid.y++;

	// launch convolution kernel with sobel matrix
	kernelMatrixConvolution<<< grid, block >>>(image.data(), image.pitch(),
											   sobelX.data(), sobelX.pitch(),
											   image.width(), image.height(),
											   sobelXFilter,
											   3, 3);

	kernelMatrixConvolution<<< grid, block >>>(image.data(), image.pitch(),
											   sobelY.data(), sobelY.pitch(),
											   image.width(), image.height(),
											   sobelYFilter,
											   3, 3);

	// vector sum of both sobel images
	kernelVectorSum<<< grid, block >>>(sobelX.data(), sobelX.pitch(),
									   sobelY.data(), sobelY.pitch(),
									   edges.data(), edges.pitch(),
									   image.width(), image.height());

	// sync and check for errors
	hipDeviceSynchronize(); cudaErr(hipGetLastError());

	// success
	return 0;
}